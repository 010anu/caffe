#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype>
__global__ void FixedUnPoolForward(const int nthreads, const Dtype* bottom_data,
    const int num, const int channels, const int height, const int width,
    const int unpooled_height, const int unpooled_width, const int out_kernel_h,
    const int out_kernel_w, const int out_stride_h, const int out_stride_w,
    const int out_pad_h, const int out_pad_w, Dtype* top_data) {
  CUDA_KERNEL_LOOP(unpool_index, nthreads) {
    int uw = unpool_index % unpooled_width;
    int uh = (unpool_index / unpooled_width) % unpooled_height;
    int c = (unpool_index / unpooled_width / unpooled_height) % channels;
    int n = unpool_index / unpooled_width / unpooled_height / channels;
    int hstart = (uh + out_pad_h < out_kernel_h) ? 0 :
      (uh + out_pad_h - out_kernel_h) / out_stride_h + 1;
    int hend = min((uh + out_pad_h) / out_stride_h + 1, height);
    int wstart = (uw + out_pad_w < out_kernel_w) ? 0 :
      (uw + out_pad_w - out_kernel_w) / out_stride_w + 1;
    int wend = min((uw + out_pad_w) / out_stride_w + 1, width);
    int offset = (n * channels + c) * height * width;
    int unpool_offset = (n * channels + c) * unpooled_height * unpooled_width;
    bottom_data += offset;
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        int uhstart = h * out_stride_h - out_pad_h;
        int uwstart = w * out_stride_w - out_pad_w;
        int uhend = uhstart + out_kernel_h;
        int uwend = uwstart + out_kernel_w;
        int uhmid = (uhstart + uhend - 1) / 2;
        int uwmid = (uwstart + uwend - 1) / 2;
        uhmid = min(max(uhmid, 0), unpooled_height);
        uwmid = min(max(uwmid, 0), unpooled_width);
        if (unpool_offset + uhmid * unpooled_width + uwmid == unpool_index) {
          // find the mapping, assign & return
          int index = h * width + w;
          top_data[unpool_index] = bottom_data[index];
          return;
        }
      }
    }
  }
}

template <typename Dtype>
__global__ void DivUnPoolForward(const int nthreads, const Dtype* bottom_data,
    const int* mask, const int num, const int channels, const int height,
    const int width, const int unpooled_height, const int unpooled_width,
    const int out_kernel_h, const int out_kernel_w, const int out_stride_h,
    const int out_stride_w, const int out_pad_h, const int out_pad_w,
    Dtype* top_data) {
  CUDA_KERNEL_LOOP(unpool_index, nthreads) {
    int uw = unpool_index % unpooled_width + out_pad_w;
    int uh = (unpool_index / unpooled_width) % unpooled_height + out_pad_h;
    int c = (unpool_index / unpooled_width / unpooled_height) % channels;
    int n = unpool_index / unpooled_width / unpooled_height / channels;
    int spatial_dim = unpooled_height * unpooled_width;
    int hstart = (uh < out_kernel_h) ? 0 :
      (uh - out_kernel_h) / out_stride_h + 1;
    int hend = min(uh / out_stride_h + 1, height);
    int wstart = (uw < out_kernel_w) ? 0 : 
      (uw - out_kernel_w) / out_stride_w + 1;
    int wend = min(uw / out_stride_w + 1, width);
    Dtype divval = 0;
    bottom_data += (n * channels + c) * height * width;
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        int uhstart = h * out_stride_h - out_pad_h;
        int uwstart = w * out_stride_w - out_pad_w;
        int uhend = min(uhstart + out_kernel_h, unpooled_height + out_pad_h);
        int uwend = min(uwstart + out_kernel_w, unpooled_width + out_pad_w);
        int unpool_size = (uhend - uhstart) * (uwend - uwstart);
        divval += bottom_data[h * width + w] / unpool_size;
      }
    }
    top_data[unpool_index] = divval / mask[unpool_index % spatial_dim];
  }
}

template <typename Dtype>
__global__ void RepUnPoolForward(const int nthreads, const Dtype* bottom_data,
    const int* mask, const int num, const int channels, const int height,
    const int width, const int unpooled_height, const int unpooled_width,
    const int out_kernel_h, const int out_kernel_w, const int out_stride_h,
    const int out_stride_w, const int out_pad_h, const int out_pad_w,
    Dtype* top_data) {
  CUDA_KERNEL_LOOP(unpool_index, nthreads) {
    int uw = unpool_index % unpooled_width + out_pad_w;
    int uh = (unpool_index / unpooled_width) % unpooled_height + out_pad_h;
    int c = (unpool_index / unpooled_width / unpooled_height) % channels;
    int n = unpool_index / unpooled_width / unpooled_height / channels;
    int spatial_dim = unpooled_height * unpooled_width;
    int hstart = (uh < out_kernel_h) ? 0 :
      (uh - out_kernel_h) / out_stride_h + 1;
    int hend = min(uh / out_stride_h + 1, height);
    int wstart = (uw < out_kernel_w) ? 0 : 
      (uw - out_kernel_w) / out_stride_w + 1;
    int wend = min(uw / out_stride_w + 1, width);
    Dtype val = 0;
    bottom_data += (n * channels + c) * height * width;
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        int uhstart = h * out_stride_h - out_pad_h;
        int uwstart = w * out_stride_w - out_pad_w;
        int uhend = min(uhstart + out_kernel_h, unpooled_height + out_pad_h);
        int uwend = min(uwstart + out_kernel_w, unpooled_width + out_pad_w);
        val += bottom_data[h * width + w];
      }
    }
    top_data[unpool_index] = val / mask[unpool_index % spatial_dim];
  }
}

// convert map to Blob because CUDA code cannot access map STL
void GetMapData(const map<int, vector<int> >& group_map,
    Blob<int>* group_map_range, Blob<int>* group_map_index = NULL) {
  // get the start and end index of all groups
  group_map_range->Reshape(1, 1, 1, group_map.size()+1);
  int* group_map_range_data = group_map_range->mutable_cpu_data();
  int total_count = 0;
  int count = 0;
  for (map<int, vector<int> >::const_iterator it = group_map.begin();
      it != group_map.end(); ++it) {
    group_map_range_data[count] = total_count;
    total_count += it->second.size();
    ++count;
  }
  group_map_range_data[count] = total_count;
  // get the group_map_index if necessary
  if (group_map_index != NULL) {
    group_map_index->Reshape(1, 1, 1, total_count);
    int* group_map_index_data = group_map_index->mutable_cpu_data();
    count = 0;
    for (map<int, vector<int> >::const_iterator it = group_map.begin();
        it != group_map.end(); ++it) {
      for (int s = 0; s < it->second.size(); ++s) {
        group_map_index_data[count] = it->second[s];
        ++count;
      }
    }
  }
}

template <typename Dtype>
__global__ void ComputeGroupMean(const int nthreads, const Dtype* data,
    const int channels, const int height, const int width, const int num_groups,
    const int* group_map_range, const int* group_map_index,
    const int group_channels, Dtype* group_mean_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int group_id = index % num_groups;
    int c = (index / num_groups) % channels;
    data += c * height * width;
    int start_idx = group_map_range[group_id];
    int end_idx = group_map_range[group_id + 1];
    Dtype sumval = 0;
    for (int i = start_idx; i < end_idx; ++i) {
      sumval += data[group_map_index[i]];
    }
    group_mean_data[index] = sumval / (end_idx - start_idx) / group_channels;
  }
}

template <typename Dtype>
__global__ void GroupUnPoolForward(const int nthreads, const Dtype* group_mean_data,
    const int channels, const int height, const int width, const Dtype* group_data,
    const int num_groups, Dtype* top_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int w = index % width;
    int h = (index / width) % height;
    int c = (index / width / height) % channels;
    int group_id = (int)group_data[h * width + w];
    top_data[index] += group_mean_data[c * num_groups + group_id];
  }
}

template <typename Dtype>
void UnPoolingLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  const Dtype* group_data = NULL;
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int count = top[0]->count();
  caffe_gpu_set(count, Dtype(0), top_data);
  const int bottom_count = bottom[0]->count() / num_;
  const int* mask = mask_.gpu_data();
  switch (this->layer_param_.unpooling_param().unpool()) {
  case UnPoolingParameter_UnPoolMethod_FIXED:
    // NOLINT_NEXT_LINE(whitespace/operators)
    FixedUnPoolForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, bottom_data, num_, channels_, height_, width_,
        unpooled_height_, unpooled_width_, out_kernel_h_, out_kernel_w_,
        out_stride_h_, out_stride_w_, out_pad_h_, out_pad_w_, top_data);
    break;
  case UnPoolingParameter_UnPoolMethod_DIV:
    // NOLINT_NEXT_LINE(whitespace/operators)
    DivUnPoolForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, bottom_data, mask, num_, channels_, height_, width_,
        unpooled_height_, unpooled_width_, out_kernel_h_, out_kernel_w_,
        out_stride_h_, out_stride_w_, out_pad_h_, out_pad_w_, top_data);
    break;
  case UnPoolingParameter_UnPoolMethod_REP:
    // NOLINT_NEXT_LINE(whitespace/operators)
    RepUnPoolForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, bottom_data, mask, num_, channels_, height_, width_,
        unpooled_height_, unpooled_width_, out_kernel_h_, out_kernel_w_,
        out_stride_h_, out_stride_w_, out_pad_h_, out_pad_w_, top_data);
    break;
  case UnPoolingParameter_UnPoolMethod_GROUP:
    // NOLINT_NEXT_LINE(whitespace/operators)
    CHECK_EQ(bottom.size(), 2);
    // use the reordered internal group data
    group_data = group_blob_.gpu_data();
    for (int n = 0; n < num_; ++n) {
      const vector<map<int, vector<int> > >& group_maps = group_maps_vec_[n];
      for (int gc = 0; gc < group_channels_; ++gc) {
        const map<int, vector<int> >& group_map = group_maps[gc];
        const int num_groups = group_map.size();
        // cuda function cannot call STL function, convert it to Blob data
        GetMapData(group_map, &group_map_range_, &group_map_index_);
        const int* group_map_range = group_map_range_.gpu_data();
        const int* group_map_index = group_map_index_.gpu_data();
        // prepare group_mean_
        group_mean_.Reshape(1, channels_, 1, num_groups);
        Dtype* group_mean_data = group_mean_.mutable_gpu_data();
        int group_count = group_mean_.count();
        // compute group_mean_data
        ComputeGroupMean<Dtype><<<CAFFE_GET_BLOCKS(group_count), CAFFE_CUDA_NUM_THREADS>>>(
            group_count, bottom_data, channels_, height_, width_, num_groups,
            group_map_range, group_map_index, group_channels_, group_mean_data);
        // spread group_mean_data to top_data
        GroupUnPoolForward<Dtype><<<CAFFE_GET_BLOCKS(bottom_count), CAFFE_CUDA_NUM_THREADS>>>(
            bottom_count, group_mean_data, channels_, height_, width_,
            group_data, num_groups, top_data);
        group_data += bottom[1]->offset(0, 1);
      }
      bottom_data += bottom[0]->offset(1);
      top_data += top[0]->offset(1);
    }
    break;
  default:
    LOG(FATAL) << "Unknown unpooling method.";
  }
  CUDA_POST_KERNEL_CHECK;
}


template <typename Dtype>
__global__ void FixedUnPoolBackward(const int nthreads, const Dtype* top_diff,
    const int* mask, const int num, const int channels, const int height,
    const int width, const int unpooled_height, const int unpooled_width,
    const int out_kernel_h, const int out_kernel_w, const int out_stride_h,
    const int out_stride_w, const int out_pad_h, const int out_pad_w,
    Dtype* bottom_diff) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    // find out the local index
    // find out the local offset
    int w = index % width;
    int h = (index / width) % height;
    int c = (index / width / height) % channels;
    int n = index / width / height / channels;
    int uhstart = h * out_stride_h - out_pad_h;
    int uwstart = w * out_stride_w - out_pad_w;
    int uhend = uhstart + out_kernel_h;
    int uwend = uwstart + out_kernel_w;
    int uhmid = (uhstart + uhend - 1) / 2;
    int uwmid = (uwstart + uwend - 1) / 2;
    uhmid = min(max(uhmid, 0), unpooled_height-1);
    uwmid = min(max(uwmid, 0), unpooled_width-1);
    int offset = (n * channels + c) * unpooled_height * unpooled_width;
    int unpool_index = uhmid * unpooled_width + uwmid;
    Dtype gradient = 0;
    if (mask[unpool_index] == h * width + w) {
      gradient += top_diff[unpool_index + offset];
    }
    bottom_diff[index] = gradient;
  }
}

template <typename Dtype>
__global__ void DivUnPoolBackward(const int nthreads, const Dtype* top_diff,
    const int* mask, const int num, const int channels, const int height,
    const int width, const int unpooled_height, const int unpooled_width,
    const int out_kernel_h, const int out_kernel_w, const int out_stride_h,
    const int out_stride_w, const int out_pad_h, const int out_pad_w,
    Dtype* bottom_diff) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    // find out the local index
    // find out the local offset
    int w = index % width;
    int h = (index / width) % height;
    int c = (index / width / height) % channels;
    int n = index / width / height / channels;
    int uhstart = h * out_stride_h - out_pad_h;
    int uwstart = w * out_stride_w - out_pad_w;
    int uhend = min(uhstart + out_kernel_h, unpooled_height + out_pad_h);
    int uwend = min(uwstart + out_kernel_w, unpooled_width + out_pad_w);
    int unpool_size = (uhend - uhstart) * (uwend - uwstart);
    uhstart = max(uhstart, 0);
    uwstart = max(uwstart, 0);
    uhend = min(uhend, unpooled_height);
    uwend = min(uwend, unpooled_width);
    Dtype gradient = 0;
    int offset = (n * channels + c) * unpooled_height * unpooled_width;
    for (int uh = uhstart; uh < uhend; ++uh) {
      for (int uw = uwstart; uw < uwend; ++uw) {
        int unpool_index = uh * unpooled_width + uw;
        gradient += top_diff[unpool_index + offset] / mask[unpool_index];
      }
    }
    bottom_diff[index] = gradient / unpool_size;
  }
}

template <typename Dtype>
__global__ void RepUnPoolBackward(const int nthreads, const Dtype* top_diff,
    const int* mask, const int num, const int channels, const int height,
    const int width, const int unpooled_height, const int unpooled_width,
    const int out_kernel_h, const int out_kernel_w, const int out_stride_h,
    const int out_stride_w, const int out_pad_h, const int out_pad_w,
    Dtype* bottom_diff) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    // find out the local index
    // find out the local offset
    int w = index % width;
    int h = (index / width) % height;
    int c = (index / width / height) % channels;
    int n = index / width / height / channels;
    int uhstart = h * out_stride_h - out_pad_h;
    int uwstart = w * out_stride_w - out_pad_w;
    int uhend = min(uhstart + out_kernel_h, unpooled_height + out_pad_h);
    int uwend = min(uwstart + out_kernel_w, unpooled_width + out_pad_w);
    uhstart = max(uhstart, 0);
    uwstart = max(uwstart, 0);
    uhend = min(uhend, unpooled_height);
    uwend = min(uwend, unpooled_width);
    Dtype gradient = 0;
    int offset = (n * channels + c) * unpooled_height * unpooled_width;
    for (int uh = uhstart; uh < uhend; ++uh) {
      for (int uw = uwstart; uw < uwend; ++uw) {
        int unpool_index = uh * unpooled_width + uw;
        gradient += top_diff[unpool_index + offset] / mask[unpool_index];
      }
    }
    bottom_diff[index] = gradient;
  }
}

template <typename Dtype>
__global__ void GroupUnPoolBackward(const int nthreads, const Dtype* group_mean_diff,
    const int channels, const int height, const int width, const Dtype* group_data,
    const int num_groups, Dtype* bottom_diff) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int w = index % width;
    int h = (index / width) % height;
    int c = (index / width / height) % channels;
    int group_id = (int)group_data[h * width + w];
    bottom_diff[index] += group_mean_diff[c * num_groups + group_id];
  }
}

template <typename Dtype>
void UnPoolingLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (!propagate_down[0]) {
    return;
  }
  const Dtype* top_diff = top[0]->gpu_diff();
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
  const Dtype* group_data;
  const int count = bottom[0]->count();
  const int top_count = top[0]->count() / top[0]->num();
  caffe_gpu_set(count, Dtype(0.), bottom_diff);
  const int* mask = mask_.gpu_data();
  switch (this->layer_param_.unpooling_param().unpool()) {
  case UnPoolingParameter_UnPoolMethod_FIXED:
    // NOLINT_NEXT_LINE(whitespace/operators)
    FixedUnPoolBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, top_diff, mask, num_, channels_, height_, width_,
        unpooled_height_, unpooled_width_, out_kernel_h_, out_kernel_w_,
        out_stride_h_, out_stride_w_, out_pad_h_, out_pad_w_, bottom_diff);
    break;
  case UnPoolingParameter_UnPoolMethod_DIV:
    // NOLINT_NEXT_LINE(whitespace/operators)
    DivUnPoolBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, top_diff, mask, num_, channels_, height_, width_,
        unpooled_height_, unpooled_width_, out_kernel_h_, out_kernel_w_,
        out_stride_h_, out_stride_w_, out_pad_h_, out_pad_w_, bottom_diff);
    break;
  case UnPoolingParameter_UnPoolMethod_REP:
    // NOLINT_NEXT_LINE(whitespace/operators)
    RepUnPoolBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, top_diff, mask, num_, channels_, height_, width_,
        unpooled_height_, unpooled_width_, out_kernel_h_, out_kernel_w_,
        out_stride_h_, out_stride_w_, out_pad_h_, out_pad_w_, bottom_diff);
    break;
  case UnPoolingParameter_UnPoolMethod_GROUP:
    // NOLINT_NEXT_LINE(whitespace/operators)
    CHECK_EQ(bottom.size(), 2);
    // use the reordered internal group data
    group_data = group_blob_.gpu_data();
    for (int n = 0; n < num_; ++n) {
      const vector<map<int, vector<int> > >& group_maps = group_maps_vec_[n];
      for (int gc = 0; gc < group_channels_; ++gc) {
        const map<int, vector<int> >& group_map = group_maps[gc];
        const int num_groups = group_map.size();
        // cuda function cannot call STL function, convert it to Blob data
        GetMapData(group_map, &group_map_range_, &group_map_index_);
        const int* group_map_range = group_map_range_.gpu_data();
        const int* group_map_index = group_map_index_.gpu_data();
        // prepare group_mean_
        group_mean_.Reshape(1, channels_, 1, num_groups);
        Dtype* group_mean_diff = group_mean_.mutable_gpu_diff();
        int group_count = group_mean_.count();
        // compute group_mean_diff
        ComputeGroupMean<Dtype><<<CAFFE_GET_BLOCKS(group_count), CAFFE_CUDA_NUM_THREADS>>>(
            group_count, top_diff, channels_, height_, width_, num_groups,
            group_map_range, group_map_index, group_channels_, group_mean_diff);
        // spread group_mean_diff to bottom_diff
        GroupUnPoolBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
            top_count, group_mean_diff, channels_, height_, width_, group_data,
            num_groups, bottom_diff);
        group_data += bottom[1]->offset(0, 1);
      }
      bottom_diff += bottom[0]->offset(1);
      top_diff += top[0]->offset(1);
    }
    break;
  default:
    LOG(FATAL) << "Unknown unpooling method.";
  }
  CUDA_POST_KERNEL_CHECK;
}


INSTANTIATE_LAYER_GPU_FUNCS(UnPoolingLayer);


}  // namespace caffe
