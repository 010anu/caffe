#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "thrust/device_vector.h"

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype>
void NormalizeLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  Dtype* squared_data = squared_.mutable_gpu_data();
  Dtype* norm_data;
  if (across_spatial_) {
    // need to index it
    norm_data = norm_.mutable_cpu_data();
  } else {
    norm_data = norm_.mutable_gpu_data();
    caffe_gpu_set(norm_.count(), Dtype(0), norm_data);
  }
  int num = bottom[0]->num();
  int dim = bottom[0]->count() / num;
  int spatial_dim = bottom[0]->height() * bottom[0]->width();
  int channels = bottom[0]->channels();
  // add eps to avoid overflow
  Dtype eps = 1e-10;
  for (int n = 0; n < num; ++n) {
    caffe_gpu_powx<Dtype>(dim, bottom_data, Dtype(2), squared_data);
    if (across_spatial_) {
      Dtype normsqr;
      caffe_gpu_asum<Dtype>(dim, squared_data, &normsqr);
      norm_data[n] = pow(normsqr, Dtype(0.5)) + eps;
      caffe_gpu_scale<Dtype>(dim, scale_ / norm_data[n], bottom_data, top_data);
    } else {
      for (int c = 0; c < channels; ++c) {
        caffe_gpu_add<Dtype>(spatial_dim, squared_data+c*spatial_dim, norm_data,
            norm_data);
      }
      caffe_gpu_powx<Dtype>(spatial_dim, norm_data, Dtype(0.5), norm_data);
      // add eps to avoid overflow
      caffe_gpu_add_scalar<Dtype>(spatial_dim, eps, norm_data);
      for (int c = 0; c < channels; ++c) {
        caffe_gpu_div<Dtype>(spatial_dim, bottom_data+c*spatial_dim, norm_data,
            top_data+c*spatial_dim);
      }
      if (scale_ != 1) {
        caffe_gpu_scale<Dtype>(dim, scale_, top_data, top_data);
      }
      norm_data += spatial_dim;
    }
    bottom_data += dim;
    top_data += dim;
  }
}

template <typename Dtype>
void NormalizeLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  const Dtype* top_diff = top[0]->gpu_diff();
  const Dtype* top_data = top[0]->gpu_data();
  const Dtype* bottom_data = bottom[0]->mutable_gpu_data();
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
  const Dtype* norm_data;
  if (across_spatial_) {
    // need to index it
    norm_data = norm_.cpu_data();
  } else {
    norm_data = norm_.gpu_data();
  }
  Dtype* squared_data = squared_.mutable_gpu_data();
  int num = top[0]->num();
  int dim = top[0]->count() / num;
  int spatial_dim = top[0]->height() * top[0]->width();
  int channels = top[0]->channels();
  for (int n = 0; n < num; ++n) {
    if (across_spatial_) {
      Dtype a;
      caffe_gpu_dot<Dtype>(dim, top_data, top_diff, &a);
      caffe_gpu_scale<Dtype>(dim, a / scale_ / scale_, top_data, bottom_diff);
      caffe_gpu_sub<Dtype>(dim, top_diff, bottom_diff, bottom_diff);
      caffe_gpu_scale<Dtype>(dim, scale_ / norm_data[n], bottom_diff, bottom_diff);
    } else {
      // use squared_data to store temp result
      // dot product between top_data and top_diff
      caffe_gpu_mul<Dtype>(dim, top_data, top_diff, squared_data);
      for (int c = 1; c < channels; ++c) {
        caffe_gpu_add<Dtype>(spatial_dim, squared_data+c*spatial_dim, squared_data,
            squared_data);
      }
      // scale bottom_diff
      for (int c = 0; c < channels; ++c) {
        caffe_gpu_mul<Dtype>(spatial_dim, top_data+c*spatial_dim, squared_data,
            bottom_diff+c*spatial_dim);
      }
      if (scale_ != 1) {
        caffe_gpu_scale<Dtype>(dim, 1 / scale_ / scale_, bottom_diff, bottom_diff);
      }
      caffe_gpu_sub<Dtype>(dim, top_diff, bottom_diff, bottom_diff);
      // divide by norm
      for (int c = 0; c < channels; ++c) {
        caffe_gpu_div<Dtype>(spatial_dim, bottom_diff+c*spatial_dim, norm_data,
            bottom_diff+c*spatial_dim);
      }
      if (scale_ != 1) {
        caffe_gpu_scale<Dtype>(dim, scale_, bottom_diff, bottom_diff);
      }
      norm_data += spatial_dim;
    }
    top_data += dim;
    top_diff += dim;
    bottom_diff += dim;
    bottom_data += dim;
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(NormalizeLayer);


}  // namespace caffe
