#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>
#include <limits>

#include "thrust/device_vector.h"

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype>
void NormalizeLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  Dtype* squared_data = squared_.mutable_gpu_data();
  Dtype* norm_data = norm_.mutable_gpu_data();
  caffe_gpu_set(norm_.count(), Dtype(0), norm_data);
  Dtype normsqr;
  int num = bottom[0]->num();
  int dim = bottom[0]->count() / num;
  int spatial_dim = bottom[0]->height() * bottom[0]->width();
  int channels = bottom[0]->channels();
  // add eps to avoid overflow
  Dtype eps = std::numeric_limits<Dtype>::epsilon();
  for (int n = 0; n < num; ++n) {
    caffe_gpu_powx<Dtype>(dim, bottom_data, Dtype(2), squared_data);
    if (across_spatial_) {
      caffe_gpu_asum<Dtype>(dim, squared_data, &normsqr);
      caffe_gpu_scale<Dtype>(dim, Dtype(1)/(pow(normsqr, Dtype(0.5))+eps),
          bottom_data, top_data);
    } else {
      for (int c = 0; c < channels; ++c) {
        caffe_gpu_add<Dtype>(spatial_dim, squared_data+c*spatial_dim, norm_data,
            norm_data);
      }
      caffe_gpu_powx<Dtype>(spatial_dim, norm_data, Dtype(0.5), norm_data);
      for (int c = 0; c < channels; ++c) {
        caffe_gpu_div<Dtype>(spatial_dim, bottom_data+c*spatial_dim, norm_data,
            top_data+c*spatial_dim);
      }
      norm_data += spatial_dim;
    }
    bottom_data += dim;
    top_data += dim;
  }
}

template <typename Dtype>
void NormalizeLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  const Dtype* top_diff = top[0]->gpu_diff();
  const Dtype* top_data = top[0]->gpu_data();
  const Dtype* bottom_data = bottom[0]->mutable_gpu_data();
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
  const Dtype* norm_data = norm_.gpu_data();
  Dtype* squared_data = squared_.mutable_gpu_data();
  int num = top[0]->num();
  int dim = top[0]->count() / num;
  int spatial_dim = top[0]->height() * top[0]->width();
  int channels = top[0]->channels();
  Dtype eps = std::numeric_limits<Dtype>::epsilon();
  for (int n = 0; n < num; ++n) {
    if (across_spatial_) {
      Dtype a;
      caffe_gpu_dot<Dtype>(dim, top_data, top_diff, &a);
      caffe_gpu_scale<Dtype>(dim, a, top_data, bottom_diff);
      caffe_gpu_sub<Dtype>(dim, top_diff, bottom_diff, bottom_diff);
      caffe_gpu_dot<Dtype>(dim, bottom_data, bottom_data, &a);
      caffe_gpu_scale<Dtype>(dim, Dtype(1)/(pow(a,Dtype(0.5))+eps), bottom_diff,
          bottom_diff);
    } else {
      // use squared_data to store temp result
      // dot product between top_data and top_diff
      caffe_gpu_mul<Dtype>(dim, top_data, top_diff, squared_data);
      for (int c = 1; c < channels; ++c) {
        caffe_gpu_add<Dtype>(spatial_dim, squared_data+c*spatial_dim, squared_data,
            squared_data);
      }
      // scale bottom_diff
      for (int c = 0; c < channels; ++c) {
        caffe_gpu_mul<Dtype>(spatial_dim, top_data+c*spatial_dim, squared_data,
            bottom_diff+c*spatial_dim);
      }
      caffe_gpu_sub<Dtype>(dim, top_diff, bottom_diff, bottom_diff);
      // divide by norm
      for (int c = 0; c < channels; ++c) {
        caffe_gpu_div<Dtype>(spatial_dim, bottom_diff+c*spatial_dim, norm_data,
            bottom_diff+c*spatial_dim);
      }
      norm_data += spatial_dim;
    }
    top_data += dim;
    top_diff += dim;
    bottom_diff += dim;
    bottom_data += dim;
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(NormalizeLayer);


}  // namespace caffe
