#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "thrust/device_vector.h"

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype>
void NormalizeLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  Dtype* squared_data = squared_.mutable_gpu_data();
  Dtype* norm_data;
  if (across_spatial_) {
    // need to index it
    norm_data = norm_.mutable_cpu_data();
  } else {
    norm_data = norm_.mutable_gpu_data();
    caffe_gpu_set(norm_.count(), Dtype(0), norm_data);
  }
  const Dtype* scale = this->blobs_[0]->cpu_data();
  int num = bottom[0]->num();
  int dim = bottom[0]->count() / num;
  int spatial_dim = bottom[0]->height() * bottom[0]->width();
  int channels = bottom[0]->channels();
  for (int n = 0; n < num; ++n) {
    caffe_gpu_powx<Dtype>(dim, bottom_data, Dtype(2), squared_data);
    if (across_spatial_) {
      Dtype normsqr;
      caffe_gpu_asum<Dtype>(dim, squared_data, &normsqr);
      // add eps to avoid overflow
      norm_data[n] = pow(normsqr+eps_, Dtype(0.5));
      caffe_gpu_scale<Dtype>(dim, Dtype(1.0 / norm_data[n]), bottom_data, top_data);
    } else {
      for (int c = 0; c < channels; ++c) {
        caffe_gpu_add<Dtype>(spatial_dim, squared_data+c*spatial_dim, norm_data,
            norm_data);
      }
      // add eps to avoid overflow
      caffe_gpu_add_scalar<Dtype>(spatial_dim, eps_, norm_data);
      caffe_gpu_powx<Dtype>(spatial_dim, norm_data, Dtype(0.5), norm_data);
      for (int c = 0; c < channels; ++c) {
        caffe_gpu_div<Dtype>(spatial_dim, bottom_data+c*spatial_dim, norm_data,
            top_data+c*spatial_dim);
      }
      norm_data += spatial_dim;
    }
    // scale the output
    if (channel_shared_) {
      caffe_gpu_scal<Dtype>(dim, scale[0], top_data);
    } else {
      for (int c = 0; c < channels; ++c) {
        caffe_gpu_scal<Dtype>(spatial_dim, scale[c], top_data+c*spatial_dim);
      }
    }
    bottom_data += dim;
    top_data += dim;
  }
}

template <typename Dtype>
void NormalizeLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  const Dtype* top_diff = top[0]->gpu_diff();
  const Dtype* top_data = top[0]->gpu_data();
  const Dtype* bottom_data = bottom[0]->mutable_gpu_data();
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
  const Dtype* scale = this->blobs_[0]->cpu_data();
  const Dtype* norm_data;
  if (across_spatial_) {
    // need to index it
    norm_data = norm_.cpu_data();
  } else {
    norm_data = norm_.gpu_data();
  }
  Dtype* squared_data = squared_.mutable_gpu_data();
  int count = top[0]->count();
  int num = top[0]->num();
  int dim = count / num;
  int spatial_dim = top[0]->height() * top[0]->width();
  int channels = top[0]->channels();

  // Propagate to param
  if (this->param_propagate_down_[0]) {
    Dtype* scale_diff = this->blobs_[0]->mutable_cpu_diff();
    Dtype a;
    if (channel_shared_) {
      caffe_gpu_dot<Dtype>(count, top_data, top_diff, &a);
      scale_diff[0] = a / scale[0];
    } else {
      caffe_set(this->blobs_[0]->count(), Dtype(0), scale_diff);
      for (int n = 0; n < num; ++n) {
        caffe_gpu_mul<Dtype>(dim, top_data+n*dim, top_diff+n*dim, squared_data);
        for (int c = 0; c < channels; ++c) {
          caffe_gpu_asum<Dtype>(spatial_dim, squared_data+c*spatial_dim, &a);
          scale_diff[c] += a / scale[c];
        }
      }
    }
  }

  // Propagate to bottom
  if (propagate_down[0]) {
    for (int n = 0; n < num; ++n) {
      if (across_spatial_) {
        Dtype a;
        caffe_gpu_dot<Dtype>(dim, bottom_data, top_diff, &a);
        caffe_gpu_scale<Dtype>(dim, a / norm_data[n] / norm_data[n],
            bottom_data, bottom_diff);
        caffe_gpu_sub<Dtype>(dim, top_diff, bottom_diff, bottom_diff);
        caffe_gpu_scale<Dtype>(dim, Dtype(1.0 / norm_data[n]), bottom_diff,
            bottom_diff);
      } else {
        // use squared_data to store temp result
        // dot product between bottom_data and top_diff
        caffe_gpu_mul<Dtype>(dim, bottom_data, top_diff, squared_data);
        for (int c = 1; c < channels; ++c) {
          caffe_gpu_add<Dtype>(spatial_dim, squared_data+c*spatial_dim,
              squared_data, squared_data);
        }
        // scale bottom_diff
        for (int c = 0; c < channels; ++c) {
          caffe_gpu_mul<Dtype>(spatial_dim, bottom_data+c*spatial_dim, squared_data,
              bottom_diff+c*spatial_dim);
        }
        // divide by square of norm
        caffe_gpu_powx<Dtype>(spatial_dim, norm_data, Dtype(2), squared_data);
        for (int c = 0; c < channels; ++c) {
          caffe_gpu_div<Dtype>(spatial_dim, bottom_diff+c*spatial_dim, squared_data,
              bottom_diff+c*spatial_dim);
        }
        caffe_gpu_sub<Dtype>(dim, top_diff, bottom_diff, bottom_diff);
        // divide by norm
        for (int c = 0; c < channels; ++c) {
          caffe_gpu_div<Dtype>(spatial_dim, bottom_diff+c*spatial_dim, norm_data,
              bottom_diff+c*spatial_dim);
        }
        norm_data += spatial_dim;
      }
      // scale the diff
      if (channel_shared_) {
        caffe_gpu_scal<Dtype>(dim, scale[0], bottom_diff);
      } else {
        for (int c = 0; c < channels; ++c) {
          caffe_gpu_scal<Dtype>(spatial_dim, scale[c], bottom_diff+c*spatial_dim);
        }
      }
      bottom_data += dim;
      top_diff += dim;
      bottom_diff += dim;
    }
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(NormalizeLayer);


}  // namespace caffe
